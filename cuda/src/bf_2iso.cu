#include "hip/hip_runtime.h"
#include <string.h>
#include <math.h>
#include <stdio.h>

#include "cucheck_macros.cuh"
#include "device_functions.cuh"
#include "create_grid.cuh"
#include "bf_2iso.cuh"

/*
Kernel function to calculate the pure histograms for the 2 point isotropic correlation function. 
This version does NOT considers boudary periodic conditions. It stores the counts in the XX histogram.

args:
XX: (double*) The histogram where the distances are counted.
elements: (PointW3D*) Array of the points ordered coherently with the nodes.
nodeD: (DNode) Array of DNodes each of which define a node and the elements of element that correspond to that node.
nonzero_nodes: (int) Number of nonzero nodes where the points have been classificated.
bins: (int) NUmber of bins in the XY histogram.
dmax: (float) The maximum distance of interest between points.
d_max_node: (float) The maximum internodal distance.
*/
__global__ void XX2iso(
    double *XX, PointW3D *elements, int np,
    double ds, int bins, float ddmax
);

/*
Kernel function to calculate the mixed histograms for the 2 point isotropic correlation function. 
This version does NOT include boundary periodic conditions. It stores the counts in the XY histogram.

args:
XY: (double*) The histogram where the distances are counted.
elementsD: (PointW3D*) Array of the points ordered coherently with the nodes. For the data points.
nodeD: (DNode) Array of DNodes each of which define a node and the elements of element that correspond to that node. For the data points
nonzero_Dnodes: (int) Number of nonzero nodes where the points have been classificated. For the data points
elementsR: (PointW3D*) Array of the points ordered coherently with the nodes. For the random points.
nodeR: (DNode) Array of DNodes each of which define a node and the elements of element that correspond to that node. For the random points
nonzero_Rnodes: (int) Number of nonzero nodes where the points have been classificated. For the random points
bins: (int) NUmber of bins in the XY histogram.
dmax: (float) The maximum distance of interest between points.
d_max_node: (float) The maximum internodal distance.
*/
__global__ void XY2iso(
    double *XY,
    PointW3D *elementsD, int np,
    PointW3D *elementsR, int rnp,
    double ds, int bins, float ddmax
);

__global__ void XX2iso_wpips(
    double *XX,
    PointW3D *elements, int32_t *pipsD, int pips_width, int np,
    double ds, int bins, float ddmax
);

void pcf_bf_2iso(
    PointW3D *d_dataD, int32_t *d_pipsD, hipStream_t streamDD, hipEvent_t DDcopy_done, int np,
    PointW3D **d_dataR, hipStream_t *streamRR, hipEvent_t *RRcopy_done, int *rnp,
    char **histo_names, int n_randfiles, int bins, float dmax,
    int pips_width
)
{

    /* =======================================================================*/
    /* ======================  Var declaration ===============================*/
    /* =======================================================================*/

    float time_spent;
    float ddmax = dmax*dmax;
    double ds = (double)(bins)/dmax; // floor(((double)(bins)/dmax)*1000000)/1000000;
    double *DD=NULL, **RR=NULL, **DR=NULL, *d_DD=NULL, **d_RR=NULL, **d_DR=NULL;
    int threads_perblock_dim = 32;
    int  blocks_Dx, blocks_Rx;

    // GPU timmer
    hipEvent_t start_timmer, stop_timmer;
    CUCHECK(hipEventCreate(&start_timmer));
    CUCHECK(hipEventCreate(&stop_timmer));

    //This may come from parameters
    hipStream_t *streamDR;
    streamDR = (hipStream_t*)malloc(n_randfiles*sizeof(hipStream_t));
    CHECKALLOC(streamDR);
    for (int i = 0; i < n_randfiles; i++)
        CUCHECK(hipStreamCreate(&streamDR[i]));

    //Prefix that will be used to save the histograms
    char *nameDD = NULL, *nameRR = NULL, *nameDR = NULL;
    int PREFIX_LENGTH;
    if (d_pipsD == NULL)
    {
        PREFIX_LENGTH = 10;
        nameDD = (char*)malloc(PREFIX_LENGTH*sizeof(char));
        nameRR = (char*)malloc(PREFIX_LENGTH*sizeof(char));
        nameDR = (char*)malloc(PREFIX_LENGTH*sizeof(char));
        strcpy(nameDD,"DDiso_BF_");
        strcpy(nameRR,"RRiso_BF_");
        strcpy(nameDR,"DRiso_BF_");
    }
    else
    {
        PREFIX_LENGTH = 15;
        nameDD = (char*)malloc(PREFIX_LENGTH*sizeof(char));
        nameRR = (char*)malloc(PREFIX_LENGTH*sizeof(char));
        nameDR = (char*)malloc(PREFIX_LENGTH*sizeof(char));
        strcpy(nameDD,"DDiso_BF_pips_");
        strcpy(nameRR,"RRiso_BF_pips_");
        strcpy(nameDR,"DRiso_BF_pips_");
    }

    /* =======================================================================*/
    /* =======================  Memory allocation ============================*/
    /* =======================================================================*/

    // Allocate memory for the histogram as double
    DD = (double*)malloc(bins*bins*sizeof(double));
    RR = (double**)malloc(n_randfiles*sizeof(double*));
    DR = (double**)malloc(n_randfiles*sizeof(double*));
    CHECKALLOC(DD);
    CHECKALLOC(RR);
    CHECKALLOC(DR);
    for (int i = 0; i < n_randfiles; i++)
    {
        RR[i] = (double*)malloc(bins*bins*sizeof(double));
        CHECKALLOC(RR[i]);
        DR[i] = (double*)malloc(bins*bins*sizeof(double));
        CHECKALLOC(DR[i]);
    }

    d_RR = (double**)malloc(n_randfiles*sizeof(double*));
    d_DR = (double**)malloc(n_randfiles*sizeof(double*));
    CHECKALLOC(d_DR);
    CHECKALLOC(d_RR);

    CUCHECK(hipMalloc(&d_DD, bins*bins*sizeof(double)));
    CUCHECK(hipMemsetAsync(d_DD, 0, bins*bins*sizeof(double), streamDD));
    for (int i = 0; i < n_randfiles; i++)
    {
        CUCHECK(hipMalloc(&d_RR[i], bins*bins*sizeof(double)));
        CUCHECK(hipMalloc(&d_DR[i], bins*bins*sizeof(double)));
        //Restarts the main histograms in device to zero
        CUCHECK(hipMemsetAsync(d_RR[i], 0, bins*bins*sizeof(double), streamRR[i]));
        CUCHECK(hipMemsetAsync(d_DR[i], 0, bins*bins*sizeof(double), streamDR[i]));
    }

    /* =======================================================================*/
    /* ====================== Starts kernel Launches  ========================*/
    /* =======================================================================*/

    //Compute the dimensions of the GPU grid
    //One thread for each node
    
    blocks_Dx = (int)(ceil((float)((float)(np)/(float)(threads_perblock_dim))));
    dim3 threads_perblock(threads_perblock_dim,threads_perblock_dim,1);
    dim3 gridD(blocks_Dx,blocks_Dx,1);
    
    //Dummy declaration
    dim3 gridR(2,2,1);
    dim3 gridDR(blocks_Dx,2,1);

    //Launch the kernels
    time_spent = 0; //Restarts timmer
    CUCHECK(hipEventRecord(start_timmer));
    if (d_pipsD == NULL)
        XX2iso<<<gridD,threads_perblock,0,streamDD>>>(d_DD, d_dataD, np, ds, bins, ddmax);
    else
        XX2iso_wpips<<<gridD,threads_perblock,0,streamDD>>>(d_DD, d_dataD, d_pipsD, pips_width, np, ds, bins, ddmax);
    CUCHECK(hipMemcpyAsync(DD, d_DD, bins*bins*sizeof(double), hipMemcpyDeviceToHost, streamDD));
    for (int i=0; i<n_randfiles; i++)
    {
        //Calculates grid dim for each file
        blocks_Rx = (int)(ceil((float)((float)(rnp[i])/(float)(threads_perblock_dim))));
        gridR.x = blocks_Rx;
        gridR.y = blocks_Rx;
        XX2iso<<<gridR,threads_perblock,0,streamRR[i]>>>(d_RR[i], d_dataR[i], rnp[i], ds, bins, ddmax);
        CUCHECK(hipMemcpyAsync(RR[i], d_RR[i], bins*bins*sizeof(double), hipMemcpyDeviceToHost, streamRR[i]));
        
        gridDR.y = blocks_Rx;
        hipStreamWaitEvent(streamDR[i], DDcopy_done, 0);
        hipStreamWaitEvent(streamDR[i], RRcopy_done[i], 0);
        XY2iso<<<gridDR,threads_perblock,0,streamDR[i]>>>(d_DR[i], d_dataD, np, d_dataR[i], rnp[i], ds, bins, ddmax);
        CUCHECK(hipMemcpyAsync(DR[i], d_DR[i], bins*bins*sizeof(double), hipMemcpyDeviceToHost, streamDR[i]));
    }


    //Waits for all the kernels to complete
    CUCHECK(hipDeviceSynchronize());

    nameDD = (char*)realloc(nameDD,PREFIX_LENGTH + strlen(histo_names[0]));
    strcpy(&nameDD[PREFIX_LENGTH-1],histo_names[0]);
    save_histogram1D(nameDD, bins, DD);

    for (int i=0; i<n_randfiles; i++)
    {
        nameRR = (char*)realloc(nameRR,PREFIX_LENGTH + strlen(histo_names[i+1]));
        strcpy(&nameRR[PREFIX_LENGTH-1],histo_names[i+1]);
        save_histogram1D(nameRR, bins, RR[i]);

        nameDR = (char*)realloc(nameDR,PREFIX_LENGTH + strlen(histo_names[i+1]));
        strcpy(&nameDR[PREFIX_LENGTH-1],histo_names[i+1]);
        save_histogram1D(nameDR, bins, DR[i]);
    }

    CUCHECK(hipEventRecord(stop_timmer));
    CUCHECK(hipEventSynchronize(stop_timmer));
    CUCHECK(hipEventElapsedTime(&time_spent, start_timmer, stop_timmer));

    printf("Spent %f miliseconds to compute and save all the histograms. \n", time_spent);
    
    /* =======================================================================*/
    /* ==========================  Free memory ===============================*/
    /* =======================================================================*/

    //Free the memory
    CUCHECK(hipStreamDestroy(streamDD));
    for (int i = 0; i < n_randfiles; i++)
    {
        CUCHECK(hipStreamDestroy(streamDR[i]));
        CUCHECK(hipStreamDestroy(streamRR[i]));
        
        free(RR[i]);
        RR[i] = NULL;
        free(DR[i]);
        DR[i] = NULL;
        CUCHECK(hipFree(d_RR[i]));
        d_RR[i] = NULL;
        CUCHECK(hipFree(d_DR[i]));
        d_DR[i] = NULL;
    }
    free(streamDR);
    streamDR = NULL;
    free(streamRR);
    streamRR = NULL;
    free(RR);
    RR = NULL;
    free(DR);
    DR = NULL;
    free(d_RR);
    d_RR = NULL;
    free(d_DR);
    d_DR = NULL;

    CUCHECK(hipEventDestroy(start_timmer));
    CUCHECK(hipEventDestroy(stop_timmer));

    free(DD);
    DD = NULL;
    CUCHECK(hipFree(d_DD));
    d_DD = NULL;
}

//====================================================================
//============ Kernels Section ======================================= 
//====================================================================
__global__ void XX2iso(
    double *XX, PointW3D *elements, int np, 
    double ds, int bins, float ddmax
)
{

    //Distributes all the indexes equitatively into the n_kernelc_calls.

    int idx1 = blockIdx.x * blockDim.x + threadIdx.x;
    int idx2 = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx1<np && idx2<np && idx1 < idx2)
    {
        int bin;
        float v;
        float dx = elements[idx1].x - elements[idx2].x;
        float dy = elements[idx1].y - elements[idx2].y;
        float dz = elements[idx1].z - elements[idx2].z;
        float dd = dx*dx + dy*dy + dz*dz;

        if (dd <= ddmax && dd > 0){

            bin = (int)(sqrt(dd)*ds);
            if (bin<bins)
            {
                v = elements[idx1].w*elements[idx2].w;
                atomicAdd(&XX[bin],v);
            }
        }
    }
}

__global__ void XY2iso(
    double *XY, 
    PointW3D *elementsD, int np,
    PointW3D *elementsR, int rnp,
    double ds, int bins, float ddmax
)
{
    int idx1 = blockIdx.x * blockDim.x + threadIdx.x;
    int idx2 = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx1<np && idx2<rnp)
    {
        int bin;
        float v;
        float dx = elementsD[idx1].x - elementsR[idx2].x;
        float dy = elementsD[idx1].y - elementsR[idx2].y;
        float dz = elementsD[idx1].z - elementsR[idx2].z;
        float dd = dx*dx + dy*dy + dz*dz;

        if (dd <= ddmax)
        {
            bin = (int)(sqrt(dd)*ds);
            if (bin<bins)
            {
                v = elementsD[idx1].w*elementsR[idx2].w;
                atomicAdd(&XY[bin],v);
            }
        }
    }
}

__global__ void XX2iso_wpips(
    double *XX, PointW3D *elements, int32_t *pipsD, int pips_width, int np,
    double ds, int bins, float ddmax
)
{

    //Distributes all the indexes equitatively into the n_kernelc_calls.

    int idx1 = blockIdx.x * blockDim.x + threadIdx.x;
    int idx2 = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx1<np && idx2<np && idx1 < idx2)
    {
        int bin;
        float v;
        float dx = elements[idx1].x - elements[idx2].x;
        float dy = elements[idx1].y - elements[idx2].y;
        float dz = elements[idx1].z - elements[idx2].z;
        float dd = dx*dx + dy*dy + dz*dz;

        if (dd <= ddmax && dd > 0)
        {
            bin = (int)(sqrt(dd)*ds);
            if (bin<bins)
            {
                v = get_weight(pipsD, idx1, pipsD, idx2, pips_width);
                atomicAdd(&XX[bin],v);
            }
        }
    }
}
