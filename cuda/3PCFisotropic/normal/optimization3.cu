#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string.h>
#include <time.h>
#include <math.h>
#include "create_grid.cuh"
#include "kernels.cuh"

using namespace std;

/** CUDA check macro */
#define cucheck(call) \
	{\
	hipError_t res = (call);\
	if(res != hipSuccess) {\
	const char* err_str = hipGetErrorString(res);\
	fprintf(stderr, "%s (%d): %s in %s \n", __FILE__, __LINE__, err_str, #call);	\
	exit(-1);\
	}\
	}


int main(int argc, char **argv){
    /*
    Main function to calculate the isotropic 3 point correlation function. Saves three different histograms in the same location of this script
    with the names DD.dat DR.dat RR.dat. This program do not consider periodic boundary conditions. The file must contain 4 columns, the first 3 
    are the x,y,z coordinates and the 4 the weigh of the measurment.

    Args:
    arg[1]: name or path to the data file relative to ../../../fake_DATA/DATOS/. 
    arg[2]: name or path to the random file relative to ../../../fake_DATA/DATOS/
    arg[3]: integer of the number of points in the files.
    arg[4]: integer. Number of bins where the distances are classified
    arg[5]: float. Maximum distance of interest. It has to have the same units as the points in the files.
    */

    /* =======================================================================*/
    /* =====================   Var declaration ===============================*/
    /* =======================================================================*/

    unsigned int np = stoi(argv[3]), bn = stoi(argv[4]), partitions;

    float time_spent, d_max_node, size_node, dmax = stof(argv[5]), size_box = 0, r_size_box=0;

    double *DDD;//, *RRR, *DRR, *DDR;
    double *d_DDD;//, *d_RRR, *d_DRR, *d_DDR;

    //n_kernel_calls should depend of the number of points, its density, and the number of bins
    int threads_perblock_dim = 8, nonzero_Dnodes = 0, idxD=0;//, nonzero_Rnodes = 0, idxR=0;
    int blocks_D;//, blocks_R;

    hipEvent_t start_timmer, stop_timmer; // GPU timmer
    cucheck(hipEventCreate(&start_timmer));
    cucheck(hipEventCreate(&stop_timmer));

    clock_t stop_timmer_host, start_timmer_host;

    PointW3D *dataD;
    //PointW3D *dataR;

    int k_element, last_pointD;//, last_pointR;
    Node ***hnodeD;//, ***hnodeR;
    DNode *hnodeD_s;//, *hnodeR_s;
    PointW3D *h_ordered_pointsD_s;//, *h_ordered_pointsR_s;

    hipStream_t streamDDD;//, streamDRR, streamDDR, streamRRR;
    cucheck(hipStreamCreate(&streamDDD));
    //cucheck(hipStreamCreate(&streamDDR));
    //cucheck(hipStreamCreate(&streamDRR));
    //cucheck(hipStreamCreate(&streamRRR));
    DNode *dnodeD_DDD;//, *dnodeD_DDR, *dnodeD_DRR;
    //DNode *dnodeR_RRR, *dnodeR_DDR, *dnodeR_DRR;
    PointW3D *d_ordered_pointsD_DDD;//, *d_ordered_pointsD_DDR, *d_ordered_pointsD_DRR;
    //PointW3D *d_ordered_pointsR_RRR, *d_ordered_pointsR_DDR, *d_ordered_pointsR_DRR;

    // Name of the files where the results are saved
    string nameDDD = "DDDiso.dat";//, nameRRR = "RRRiso.dat", nameDDR = "DDRiso.dat", nameDRR = "DRRiso.dat";

    /* =======================================================================*/
    /* =======================  Memory allocation ============================*/
    /* =======================================================================*/
    start_timmer_host = clock();
    dataD = new PointW3D[np];
    //dataR = new PointW3D[np];

    // Open and read the files to store the data in the arrays
    open_files(argv[1], np, dataD, size_box); //This function also gets the real size of the box
    //open_files(argv[2], np, dataR, r_size_box);
    if (r_size_box>size_box){
        size_box=r_size_box;
    }

    if (argc>6){
        r_size_box = stof(argv[6]);
        if (r_size_box>0){
            size_box=r_size_box;
        }
    }

    //Sets the number of partitions of the box and the size of each node
    if (argc>7){
        //Partitions entered by the user
        partitions = stof(argv[7]);
    } else {
        //Calculate optimum partitions
        partitions = 35;
    }
    size_node = size_box/(float)(partitions);
    d_max_node = dmax + size_node*sqrt(3.0);
    d_max_node*=d_max_node;

    // Allocate memory for the histogram as double
    DDD = new double[bn*bn*bn];
    //RRR = new double[bn*bn*bn];
    //DDR = new double[bn*bn*bn];
    //DRR = new double[bn*bn*bn];

    cucheck(hipMalloc(&d_DDD, bn*bn*bn*sizeof(double)));
    //cucheck(hipMalloc(&d_RRR, bn*bn*bn*sizeof(double)));
    //cucheck(hipMalloc(&d_DRR, bn*bn*bn*sizeof(double)));
    //cucheck(hipMalloc(&d_DDR, bn*bn*bn*sizeof(double)));

    //Restarts the main histograms in host to zero
    cucheck(hipMemsetAsync(d_DDD, 0, bn*bn*bn*sizeof(double), streamDDD));
    //cucheck(hipMemsetAsync(d_RRR, 0, bn*bn*bn*sizeof(double), streamRRR));
    //cucheck(hipMemsetAsync(d_DRR, 0, bn*bn*bn*sizeof(double), streamDRR));
    //cucheck(hipMemsetAsync(d_DDR, 0, bn*bn*bn*sizeof(double), streamDDR));

    hnodeD = new Node**[partitions];
    //hnodeR = new Node**[partitions];
    for (int i=0; i<partitions; i++){
        *(hnodeD+i) = new Node*[partitions];
        //*(hnodeR+i) = new Node*[partitions];
        for (int j=0; j<partitions; j++){
            *(*(hnodeD+i)+j) = new Node[partitions];
            //*(*(hnodeR+i)+j) = new Node[partitions];
        }
    }

    //Classificate the data into the nodes in the host side
    //The node classification is made in the host
    make_nodos(hnodeD, dataD, partitions, size_node, np);
    //make_nodos(hnodeR, dataR, partitions, size_node, np);

    for(int row=0; row<partitions; row++){
        for(int col=0; col<partitions; col++){
            for(int mom=0; mom<partitions; mom++){
                if(hnodeD[row][col][mom].len>0){
                    nonzero_Dnodes+=1;
                }
                //if(hnodeR[row][col][mom].len>0){
                    //nonzero_Rnodes+=1;
                //}
            }
        }
    }

    //Allocate memory for the nodes depending of how many partitions there are.
    cucheck(hipMalloc(&dnodeD_DDD, nonzero_Dnodes*sizeof(DNode)));
    cucheck(hipMalloc(&d_ordered_pointsD_DDD, np*sizeof(PointW3D)));
    //cucheck(hipMalloc(&dnodeD_DDR, nonzero_Dnodes*sizeof(DNode)));
    //cucheck(hipMalloc(&d_ordered_pointsD_DDR, np*sizeof(PointW3D)));
    //cucheck(hipMalloc(&dnodeD_DRR, nonzero_Dnodes*sizeof(DNode)));
    //cucheck(hipMalloc(&d_ordered_pointsD_DRR, np*sizeof(PointW3D)));

    //cucheck(hipMalloc(&dnodeR_RRR, nonzero_Rnodes*sizeof(DNode)));
    //cucheck(hipMalloc(&d_ordered_pointsR_RRR, np*sizeof(PointW3D)));
    //cucheck(hipMalloc(&dnodeR_DDR, nonzero_Rnodes*sizeof(DNode)));
    //cucheck(hipMalloc(&d_ordered_pointsR_DDR, np*sizeof(PointW3D)));
    //cucheck(hipMalloc(&dnodeR_DRR, nonzero_Rnodes*sizeof(DNode)));
    //cucheck(hipMalloc(&d_ordered_pointsR_DRR, np*sizeof(PointW3D)));

    hnodeD_s = new DNode[nonzero_Dnodes];
    h_ordered_pointsD_s = new PointW3D[np];
    //hnodeR_s = new DNode[nonzero_Rnodes];
    //h_ordered_pointsR_s = new PointW3D[np];
    
    //Deep copy to device memory
    last_pointD = 0;
    //last_pointR = 0;
    for(int row=0; row<partitions; row++){
        for(int col=0; col<partitions; col++){
            for(int mom=0; mom<partitions; mom++){
        
                if (hnodeD[row][col][mom].len>0){
                    hnodeD_s[idxD].nodepos = hnodeD[row][col][mom].nodepos;
                    hnodeD_s[idxD].start = last_pointD;
                    hnodeD_s[idxD].len = hnodeD[row][col][mom].len;
                    last_pointD = last_pointD + hnodeD[row][col][mom].len;
                    hnodeD_s[idxD].end = last_pointD;
                    for (int j=hnodeD_s[idxD].start; j<last_pointD; j++){
                        k_element = j-hnodeD_s[idxD].start;
                        h_ordered_pointsD_s[j] = hnodeD[row][col][mom].elements[k_element];
                    }
                    idxD++;
                }

                // if (hnodeR[row][col][mom].len>0){
                //     hnodeR_s[idxR].nodepos = hnodeR[row][col][mom].nodepos;
                //     hnodeR_s[idxR].start = last_pointR;
                //     hnodeR_s[idxR].len = hnodeR[row][col][mom].len;
                //     last_pointR = last_pointR + hnodeR[row][col][mom].len;
                //     hnodeR_s[idxR].end = last_pointR;
                //     for (int j=hnodeR_s[idxR].start; j<last_pointR; j++){
                //         k_element = j-hnodeR_s[idxR].start;
                //         h_ordered_pointsR_s[j] = hnodeR[row][col][mom].elements[k_element];
                //     }
                //     idxR++;
                // }

            }
        }
    }


    cucheck(hipMemcpyAsync(dnodeD_DDD, hnodeD_s, nonzero_Dnodes*sizeof(DNode), hipMemcpyHostToDevice, streamDDD));
    cucheck(hipMemcpyAsync(d_ordered_pointsD_DDD, h_ordered_pointsD_s, np*sizeof(PointW3D), hipMemcpyHostToDevice, streamDDD));
    //cucheck(hipMemcpyAsync(dnodeD_DDR, hnodeD_s, nonzero_Dnodes*sizeof(DNode), hipMemcpyHostToDevice, streamDDR));
    //cucheck(hipMemcpyAsync(d_ordered_pointsD_DDR, h_ordered_pointsD_s, np*sizeof(PointW3D), hipMemcpyHostToDevice, streamDDR));
    //cucheck(hipMemcpyAsync(dnodeD_DRR, hnodeD_s, nonzero_Dnodes*sizeof(DNode), hipMemcpyHostToDevice, streamDRR));
    //cucheck(hipMemcpyAsync(d_ordered_pointsD_DRR, h_ordered_pointsD_s, np*sizeof(PointW3D), hipMemcpyHostToDevice, streamDRR));
    
    //cucheck(hipMemcpyAsync(dnodeR_RRR, hnodeR_s, nonzero_Rnodes*sizeof(DNode), hipMemcpyHostToDevice, streamRRR));
    //cucheck(hipMemcpyAsync(d_ordered_pointsR_RRR, h_ordered_pointsR_s, np*sizeof(PointW3D), hipMemcpyHostToDevice, streamRRR));
    //cucheck(hipMemcpyAsync(dnodeR_DDR, hnodeR_s, nonzero_Rnodes*sizeof(DNode), hipMemcpyHostToDevice, streamDDR));
    //cucheck(hipMemcpyAsync(d_ordered_pointsR_DDR, h_ordered_pointsR_s, np*sizeof(PointW3D), hipMemcpyHostToDevice, streamDDR));
    //cucheck(hipMemcpyAsync(dnodeR_DRR, hnodeR_s, nonzero_Rnodes*sizeof(DNode), hipMemcpyHostToDevice, streamDRR));
    //cucheck(hipMemcpyAsync(d_ordered_pointsR_DRR, h_ordered_pointsR_s, np*sizeof(PointW3D), hipMemcpyHostToDevice, streamDRR));

    //stop_timmer_host = clock();
    //time_spent = ((float)(stop_timmer_host-start_timmer_host))/CLOCKS_PER_SEC;
    //cout << "Succesfully readed the data. All set to compute the histograms in " << time_spent*1000 << " miliseconds" << endl;


    /* =======================================================================*/
    /* ====================== Starts kernel Launches  ========================*/
    /* =======================================================================*/


    //Compute the dimensions of the GPU grid
    //One thread for each node
    
    blocks_D = (int)(ceil((float)((float)(nonzero_Dnodes)/(float)(threads_perblock_dim))));
    //blocks_R = (int)(ceil((float)((float)(nonzero_Rnodes)/(float)(threads_perblock_dim))));

    dim3 threads_perblock(threads_perblock_dim,threads_perblock_dim,threads_perblock_dim);
    
    dim3 gridDDD(blocks_D,blocks_D,blocks_D);
    //dim3 gridRRR(blocks_R,blocks_R,blocks_R);
    //dim3 gridDDR(blocks_D,blocks_D,blocks_R);
    //dim3 gridDRR(blocks_R,blocks_R,blocks_D);

    //Launch the kernels
    time_spent=0; //Restarts timmer
    hipEventRecord(start_timmer);
    make_histoXXX<<<gridDDD,threads_perblock,0,streamDDD>>>(d_DDD, d_ordered_pointsD_DDD, dnodeD_DDD, nonzero_Dnodes, bn, dmax, d_max_node);
    //make_histoXXX<<<gridRRR,threads_perblock,0,streamRRR>>>(d_RRR, d_ordered_pointsR_RRR, dnodeR_RRR, nonzero_Rnodes, bn, dmax, d_max_node);
    //make_histoXXY<<<gridDRR,threads_perblock,0,streamDRR>>>(d_DRR, d_ordered_pointsR_DRR, dnodeR_DRR, nonzero_Rnodes, d_ordered_pointsD_DRR, dnodeD_DRR, nonzero_Dnodes, bn, dmax, d_max_node);
    //make_histoXXY<<<gridDDR,threads_perblock,0,streamDDR>>>(d_DDR, d_ordered_pointsD_DDR, dnodeD_DDR, nonzero_Dnodes, d_ordered_pointsR_DDR, dnodeR_DDR, nonzero_Rnodes, bn, dmax, d_max_node);

    cucheck(hipMemcpyAsync(DDD, d_DDD, bn*bn*bn*sizeof(double), hipMemcpyDeviceToHost, streamDDD));
    //cucheck(hipMemcpyAsync(RRR, d_RRR, bn*bn*bn*sizeof(double), hipMemcpyDeviceToHost, streamRRR));
    //cucheck(hipMemcpyAsync(DRR, d_DRR, bn*bn*bn*sizeof(double), hipMemcpyDeviceToHost, streamDRR));
    //cucheck(hipMemcpyAsync(DDR, d_DDR, bn*bn*bn*sizeof(double), hipMemcpyDeviceToHost, streamDDR));

    //Waits for all the kernels to complete
    cucheck(hipStreamSynchronize(streamDDD));
    //save_histogram(nameDDD, bn, DDD);
    //cucheck(hipStreamSynchronize(streamRRR));
    //save_histogram(nameRRR, bn, RRR);
    //cucheck(hipStreamSynchronize(streamDRR));
    //save_histogram(nameDRR, bn, DRR);
    //cucheck(hipStreamSynchronize(streamDDR));
    //save_histogram(nameDDR, bn, DDR);

    cucheck(hipEventRecord(stop_timmer));
    cucheck(hipEventSynchronize(stop_timmer));
    cucheck(hipEventElapsedTime(&time_spent, start_timmer, stop_timmer));

    //cout << "Spent "<< time_spent << " miliseconds to compute and save all the histograms." << endl;
    
    /* =======================================================================*/
    /* ==========================  Free memory ===============================*/
    /* =======================================================================*/

    //Free the memory

    for (int i=0; i<partitions; i++){
        for (int j=0; j<partitions; j++){
            delete[] hnodeD[i][j];
            //delete[] hnodeR[i][j];
        }
        delete[] hnodeD[i];
        //delete[] hnodeR[i];
    }    
    delete[] hnodeD;
    //delete[] hnodeR;

    delete[] dataD;
    //delete[] dataR;
    
    delete[] hnodeD_s;
    delete[] h_ordered_pointsD_s;
    //delete[] hnodeR_s;
    //delete[] h_ordered_pointsR_s;
    
    cucheck(hipStreamDestroy(streamDDD));
    //cucheck(hipStreamDestroy(streamDDR));
    //cucheck(hipStreamDestroy(streamDRR));
    //cucheck(hipStreamDestroy(streamRRR));

    cucheck(hipEventDestroy(start_timmer));
    cucheck(hipEventDestroy(stop_timmer));

    delete[] DDD;
    //delete[] RRR;
    //delete[] DRR;    
    //delete[] DDR;    
    
    cucheck(hipFree(d_DDD));
    //cucheck(hipFree(d_RRR));
    //cucheck(hipFree(d_DRR));
    //cucheck(hipFree(d_DDR));

    cucheck(hipFree(dnodeD_DDD));
    cucheck(hipFree(d_ordered_pointsD_DDD));
    //cucheck(hipFree(dnodeD_DDR));
    //cucheck(hipFree(d_ordered_pointsD_DDR));
    //cucheck(hipFree(dnodeD_DRR));
    //cucheck(hipFree(d_ordered_pointsD_DRR));

    //cucheck(hipFree(dnodeR_RRR));
    //cucheck(hipFree(d_ordered_pointsR_RRR));
    //cucheck(hipFree(dnodeR_DDR));
    //cucheck(hipFree(d_ordered_pointsR_DDR));
    //cucheck(hipFree(dnodeR_DRR));
    //cucheck(hipFree(d_ordered_pointsR_DRR));

    //cout << "Program terminated..." << endl;
    return 0;
}

