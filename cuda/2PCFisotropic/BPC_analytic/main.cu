#include "hip/hip_runtime.h"
// nvcc -arch=sm_75 main.cu -o par_s.out && ./par_s.out data_2GPc.dat data_2GPc.dat 3241792 20 160
// nvcc -arch=sm_75 main.cu -o par_s.out && ./par_s.out data_1GPc.dat data_1GPc.dat 405224 20 160
// nvcc -arch=sm_75 main.cu -o par_s.out && ./par_s.out data.dat rand0.dat 32768 30 150
// nvcc -arch=sm_75 main.cu -o par_s.out && ./par_s.out data_5K.dat rand0_5K.dat 5000 30 180

#include <stdio.h>
#include <iostream>
#include <string.h>
#include <time.h>
#include <math.h>
#include "create_grid.cuh"
#include "kernels.cuh"

using namespace std;

/** CUDA check macro */
#define cucheck(call) \
	{\
	hipError_t res = (call);\
	if(res != hipSuccess) {\
	const char* err_str = hipGetErrorString(res);\
	fprintf(stderr, "%s (%d): %s in %s \n", __FILE__, __LINE__, err_str, #call);	\
	exit(-1);\
	}\
	}


int main(int argc, char **argv){
    /*
    Main function to calculate the isotropic 2 point correlation function. Saves three different histograms in the same location of this script
    with the names DD.dat DR.dat RR.dat. This program do not consider periodic boundary conditions. The file must contain 4 columns, the first 3 
    are the x,y,z coordinates and the 4 the weigh of the measurment.

    Args:
    arg[1]: name or path to the data file relative to ../../../fake_DATA/DATOS/. 
    arg[2]: name or path to the random file relative to ../../../fake_DATA/DATOS/
    arg[3]: integer of the number of points in the files.
    arg[4]: integer. Number of bins where the distances are classified
    arg[5]: float. Maximum distance of interest. It has to have the same units as the points in the files.
    */

    /* =======================================================================*/
    /* =====================   Var declaration ===============================*/
    /* =======================================================================*/

    unsigned int np = stoi(argv[3]), bn = stoi(argv[4]), partitions;

    float time_spent, d_max_node, size_node, dmax = stof(argv[5]), size_box = 0, r_size_box=0;

    double *DD, *RR, *DR, *d_DD, *d_RR, *d_DR;
    double alpha1, beta1, dr; //For analytic RR

    //n_kernel_calls should depend of the number of points, its density, and the number of bins
    int  blocks_D, blocks_analytic, nonzero_Dnodes, threads_perblock_dim = 32, threads_perblock_analytic = 512, idxD=0;

    hipEvent_t start_timmer, stop_timmer; // GPU timmer
    cucheck(hipEventCreate(&start_timmer));
    cucheck(hipEventCreate(&stop_timmer));

    clock_t stop_timmer_host, start_timmer_host;

    PointW3D *dataD;

    Node ***hnodeD;
    DNode *hnodeD_s;
    PointW3D *h_ordered_pointsD_s;
    hipStream_t streamDD, streamRR, streamDR;
    cucheck(hipStreamCreate(&streamDD));
    cucheck(hipStreamCreate(&streamDR));
    cucheck(hipStreamCreate(&streamRR));
    DNode *dnodeD_DD;
    int k_element, last_pointD;
    PointW3D *d_ordered_pointsD_DD;

    // Name of the files where the results are saved
    string nameDD = "DDiso_", nameRR = "RRiso_", nameDR = "DRiso_";
    string data_name = argv[1], rand_name = argv[2];
    nameDD.append(data_name);
    nameRR.append(rand_name);
    nameDR.append(data_name);

    /* =======================================================================*/
    /* =======================  Memory allocation ============================*/
    /* =======================================================================*/
    start_timmer_host = clock();
    dataD = new PointW3D[np];

    // Open and read the files to store the data in the arrays
    open_files(data_name, np, dataD, size_box); //This function also gets the real size of the box
    if (argc>6){
        r_size_box = stof(argv[6]);
        if (r_size_box>0){
            size_box=r_size_box;
        }
    }
    //Sets the number of partitions of the box and the size of each node
    if (argc>7){
        //Partitions entered by the user
        partitions = stof(argv[7]);
    } else {
        //Calculate optimum partitions
        partitions = 35;
    }

    size_node = size_box/(float)(partitions);

    dr = (dmax/bn);
    beta1 = (np*np)/(size_box*size_box*size_box);
    alpha = 8*dr*dr*dr*(acos(0.0))*(beta1)/3;
    
    d_max_node = dmax + size_node*sqrt(3.0);
    d_max_node*=d_max_node;

    // Allocate memory for the histogram as double
    DD = new double[bn];
    RR = new double[bn];
    DR = new double[bn];

    cucheck(hipMalloc(&d_DD, bn*sizeof(double)));
    cucheck(hipMalloc(&d_RR, bn*sizeof(double)));
    cucheck(hipMalloc(&d_DR, bn*sizeof(double)));

    //Restarts the main histograms in device to zero
    cucheck(hipMemsetAsync(d_DD, 0, bn*sizeof(double), streamDD));
    cucheck(hipMemsetAsync(d_RR, 0, bn*sizeof(double), streamRR));
    cucheck(hipMemsetAsync(d_DR, 0, bn*sizeof(double), streamDR));

    hnodeD = new Node**[partitions];
    for (int i=0; i<partitions; i++){
        *(hnodeD+i) = new Node*[partitions];
        for (int j=0; j<partitions; j++){
            *(*(hnodeD+i)+j) = new Node[partitions];
        }
    }

    //Classificate the data into the nodes in the host side
    //The node classification is made in the host
    make_nodos(hnodeD, dataD, partitions, size_node, np);

    nonzero_Dnodes=0;
    for(int row=0; row<partitions; row++){
        for(int col=0; col<partitions; col++){
            for(int mom=0; mom<partitions; mom++){
                if(hnodeD[row][col][mom].len>0){
                    nonzero_Dnodes+=1;
                }
            }
        }
    }

    //Allocate memory for the nodes depending of how many partitions there are.
    cucheck(hipMalloc(&dnodeD_DD, nonzero_Dnodes*sizeof(DNode)));
    cucheck(hipMalloc(&d_ordered_pointsD_DD, np*sizeof(PointW3D)));

    hnodeD_s = new DNode[nonzero_Dnodes];
    h_ordered_pointsD_s = new PointW3D[np];

    //Deep copy to device memory
    last_pointD = 0;
    for(int row=0; row<partitions; row++){
        for(int col=0; col<partitions; col++){
            for(int mom=0; mom<partitions; mom++){
                if (hnodeD[row][col][mom].len>0){
                    hnodeD_s[idxD].nodepos = hnodeD[row][col][mom].nodepos;
                    hnodeD_s[idxD].start = last_pointD;
                    hnodeD_s[idxD].len = hnodeD[row][col][mom].len;
                    last_pointD = last_pointD + hnodeD[row][col][mom].len;
                    hnodeD_s[idxD].end = last_pointD;
                    for (int j=hnodeD_s[idxD].start; j<last_pointD; j++){
                        k_element = j-hnodeD_s[idxD].start;
                        h_ordered_pointsD_s[j] = hnodeD[row][col][mom].elements[k_element];
                    }
                    idxD++;
                }
            }
        }
    }

    cucheck(hipMemcpyAsync(d_ordered_pointsD_DD, h_ordered_pointsD_s, np*sizeof(PointW3D), hipMemcpyHostToDevice, streamDD));
    cucheck(hipMemcpyAsync(dnodeD_DD, hnodeD_s, nonzero_Dnodes*sizeof(DNode), hipMemcpyHostToDevice, streamDD));

    stop_timmer_host = clock();
    time_spent = ((float)(stop_timmer_host-start_timmer_host))/CLOCKS_PER_SEC;
    cout << "Succesfully readed the data. All set to compute the histograms in " << time_spent*1000 << " miliseconds" << endl;

    /* =======================================================================*/
    /* ====================== Starts kernel Launches  ========================*/
    /* =======================================================================*/

    //Compute the dimensions of the GPU grid
    //One thread for each node
    
    blocks_D = (int)(ceil((float)((float)(nonzero_Dnodes)/(float)(threads_perblock_dim))));
    dim3 threads_perblock_D(threads_perblock_dim,threads_perblock_dim,1);
    dim3 gridD(blocks_D,blocks_D,1);
    
    blocks_analytic = (int)(ceil((float)((float)(bn)/(float)(threads_perblock_analytic))));

    //Launch the kernels
    time_spent=0; //Restarts timmer
    hipEventRecord(start_timmer);
    make_histoXX<<<gridD,threads_perblock_D,0,streamDD>>>(d_DD, d_ordered_pointsD_DD, dnodeD_DD, nonzero_Dnodes, bn, dmax, d_max_node, size_box, size_node);
    cucheck(hipMemcpyAsync(DD, d_DD, bn*sizeof(double), hipMemcpyDeviceToHost, streamDD));
    make_histoRR<<<blocks_analytic,threads_perblock_analytic,0,streamRR>>>(d_RR, alpha, bn);
    cucheck(hipMemcpyAsync(RR, d_RR, bn*sizeof(double), hipMemcpyDeviceToHost, streamRR));
    
    cucheck(hipStreamSynchronize(streamDD));
    //make_histoDR<<<blocks_analytic,threads_perblock_analytic,0,streamDR>>>(d_DR, d_DD, dnodeD_DR, nonzero_Dnodes, d_ordered_pointsR_DR, dnodeR_DR, nonzero_Rnodes, bn, dmax, d_max_node, size_box, size_node);
    //cucheck(hipMemcpyAsync(DR, d_DR, bn*sizeof(double), hipMemcpyDeviceToHost, streamDR));
    
    //Waits for all the kernels to complete
    save_histogram(nameDD, bn, DD);
    cucheck(hipStreamSynchronize(streamRR));
    save_histogram(nameRR, bn, RR);
    cucheck(hipStreamSynchronize(streamDR));
    save_histogram(nameDR, bn, DR);

    cucheck(hipEventRecord(stop_timmer));
    cucheck(hipEventSynchronize(stop_timmer));
    cucheck(hipEventElapsedTime(&time_spent, start_timmer, stop_timmer));

    cout << "Spent "<< time_spent << " miliseconds to compute and save all the histograms." << endl;
    
    /* =======================================================================*/
    /* ==========================  Free memory ===============================*/
    /* =======================================================================*/

    //Free the memory
    cucheck(hipStreamDestroy(streamDD));
    cucheck(hipStreamDestroy(streamDR));
    cucheck(hipStreamDestroy(streamRR));

    cucheck(hipEventDestroy(start_timmer));
    cucheck(hipEventDestroy(stop_timmer));

    delete[] dataD;

    delete[] DD;
    delete[] RR;    
    delete[] DR;    
    
    cucheck(hipFree(d_DD));
    cucheck(hipFree(d_RR));
    cucheck(hipFree(d_DR));

    for (int i=0; i<partitions; i++){
        for (int j=0; j<partitions; j++){
            delete[] hnodeD[i][j];
        }
        delete[] hnodeD[i];
    }    
    delete[] hnodeD;

    cucheck(hipFree(d_ordered_pointsD_DD));
    cucheck(hipFree(dnodeD_DD));

    delete[] hnodeD_s;
    delete[] h_ordered_pointsD_s;

    cout << "Program terminated..." << endl;
    return 0;
}

