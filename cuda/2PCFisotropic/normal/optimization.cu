// nvcc simple.cu -o par_s.out && ./par_s.out data_5K.dat rand0_5K.dat 5000 30 180

// For dynamic parallelism
// nvcc -arch=sm_35 -rdc=true dynamic.cu -lcudadevrt -o par_d.out && ./par_d.out data_5K.dat rand0_5K.dat 5000 30 50

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string.h>
#include <time.h>
#include <math.h>
#include <limits>

using namespace std;

//Point with weight value. Structure

struct Point3D{
	float x;
	float y; 
	float z;
};

struct PointW3D{
    float x;
    float y; 
    float z;
    float w;
};

struct Node{
    Point3D nodepos; //Position of the node
    int len;		// Number of points in the node
    PointW3D *elements;	// Points in the node
};


void open_files(string name_file, int pts, PointW3D *datos, float &size_box){
    /* Opens the daya files. Receives the file location, number of points to read and the array of points where the data is stored */
    ifstream file;

    string mypathto_files = "../../../fake_DATA/DATOS/";
    //This creates the full path to where I have my data files
    name_file.insert(0,mypathto_files);

    file.open(name_file.c_str(), ios::in | ios::binary); //Tells the program this is a binary file using ios::binary
    if (file.fail()){
        cout << "Failed to load the file in " << name_file << endl;
        exit(1);
    }

    double candidate_size_box=0;
    double max_component;
    for ( int c = 0; c < pts; c++) //Reads line by line and stores each c line in the c PointW3D element of the array
    {
        file >> datos[c].x >> datos[c].y >> datos[c].z >> datos[c].w;

        if (datos[c].x>datos[c].y){
            if (datos[c].x>datos[c].z){
                max_component = datos[c].x;
            } else {
                max_component = datos[c].z;
            }

        } else {
            if (datos[c].y>datos[c].z){
                max_component = datos[c].y;
            } else {
                max_component = datos[c].z;
            }
        }

        if (max_component>candidate_size_box){
            candidate_size_box = max_component;
        }
    }

    size_box=ceil(candidate_size_box+1);

    file.close();
}

//====================================================================

void save_histogram(string name, int bns, double *histo){
    /* This function saves a one dimensional histogram in a file.
    Receives the name of the file, number of bins in the histogram and the histogram array
    */

    ofstream file2;
    file2.open(name.c_str(), ios::out | ios::binary);

    if (file2.fail()){
        cout << "Failed to save the the histogram in " << name << endl;
        exit(1);
    }
    for (int i = 0; i < bns; i++){
        file2 << histo[i] << endl;
    }
    file2.close();
}

//=================================================================== 
void add(PointW3D *&array, int &lon, float _x, float _y, float _z, float _w){
    /*
    This function manages adding points to an specific Node. It receives the previous array, longitude and point to add
    and updates the previous array and length with the same array with the new point at the end and adds +1 to the length +1

    It manages the memory allocation and free of the previous and new elements.
    */
    lon++;
    PointW3D *array_aux;
    hipMallocManaged(&array_aux, lon*sizeof(PointW3D)); 
    for (int i=0; i<lon-1; i++){
        array_aux[i].x = array[i].x;
        array_aux[i].y = array[i].y;
        array_aux[i].z = array[i].z;
        array_aux[i].w = array[i].w;
    }

    hipFree(array);
    array = array_aux;
    array[lon-1].x = _x;
    array[lon-1].y = _y;
    array[lon-1].z = _z;
    array[lon-1].w = _w;
}

void make_nodos(Node ***nod, PointW3D *dat, int partitions, float size_node, unsigned int np){
    /*
    This function classifies the data in the nodes

    Args
    nod: Node 3D array where the data will be classified
    dat: array of PointW3D data to be classified and stored in the nodes
    partitions: number nodes in each direction
    size_node: dimensions of a single node
    np: number of points in the dat array
    */

    int row, col, mom;

    // First allocate memory as an empty node:
    for (row=0; row<partitions; row++){
        for (col=0; col<partitions; col++){
            for (mom=0; mom<partitions; mom++){
                nod[row][col][mom].nodepos.z = ((float)(mom)*(size_node));
                nod[row][col][mom].nodepos.y = ((float)(col)*(size_node));
                nod[row][col][mom].nodepos.x = ((float)(row)*(size_node));
                nod[row][col][mom].len = 0;
                hipMallocManaged(&nod[row][col][mom].elements, sizeof(PointW3D));
            }
        }
    }

    // Classificate the ith elment of the data into a node and add that point to the node with the add function:
    for (int i=0; i<np; i++){
        row = (int)(dat[i].x/size_node);
        col = (int)(dat[i].y/size_node);
        mom = (int)(dat[i].z/size_node);
        add(nod[row][col][mom].elements, nod[row][col][mom].len, dat[i].x, dat[i].y, dat[i].z, dat[i].w);
    }
}

//====================================================================
//============ Kernels Section ======================================= 
//====================================================================

__device__ void count_distances11(float *XX, PointW3D *elements, int len, float ds, float dd_max, int sum){
    /*
    This device function counts the distances betweeen points within one node.

    Args:
    XX: The histogram where the distances are counted in
    elements:  Array of PointW3D points inside the node
    len: lenght of the elements array
    ds: number of bins divided by the maximum distance. Used to calculate the bin it should be counted at
    dd_max: The maximum distance of interest.
    */
    
    //printf("The id is: %i . The len: %i The blockid: %i \n. ", idx, len, blockIdx.x);
    int bin;
    float d, v;
    float x1, y1, z1, w1;
    float x2,y2,z2,w2;

    for (int idx=0; idx<len-1; ++idx){
        x1 = elements[idx].x;
        y1 = elements[idx].y;
        z1 = elements[idx].z;
        w1 = elements[idx].w;
        for (int j=idx+1; j<len; ++j){
            x2 = elements[j].x;
            y2 = elements[j].y;
            z2 = elements[j].z;
            w2 = elements[j].w;
            d = (x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1);
            if (d<=dd_max+1){
                bin = (int)(sqrt(d)*ds);
                v = sum*w1*w2;
                atomicAdd(&XX[bin],v);
            }
        }
    }
}

__device__ void count_distances12(float *XX, PointW3D *elements1, int len1, PointW3D *elements2, int len2, float ds, float dd_max, int sum){
    /*
    This device function counts the distances betweeen points between two different nodes.

    Args:
    XX: The histogram where the distances are counted in
    elements1:  Array of PointW3D points inside the first node
    len1: lenght of the first elements array
    elements2:  Array of PointW3D points inside the second node
    len2: lenght of the second elements array
    ds: number of bins divided by the maximum distance. Used to calculate the bin it should be counted at
    dd_max: The maximum distance of interest.
    */

    int bin;
    float d, v;
    float x1,y1,z1,w1,x2,y2,z2,w2;

    for (int i=0; i<len1; ++i){
        x1 = elements1[i].x;
        y1 = elements1[i].y;
        z1 = elements1[i].z;
        w1 = elements1[i].w;
        for (int j=0; j<len2; ++j){
            x2 = elements2[j].x;
            y2 = elements2[j].y;
            z2 = elements2[j].z;
            w2 = elements2[j].w;
            d = (x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1);
            if (d<=dd_max+1){
                bin = (int)(sqrt(d)*ds);
                v = sum*w1*w2;
                atomicAdd(&XX[bin],v);
            }
        }
    }
}

__global__ void make_histoXX(float *XX, Node ***nodeD, int partitions, int bn, float dmax, float size_node, int start_at){
    //If start at is 0 it does every even index, it does every odd index otherwise
    int idx = 2*(blockIdx.x * blockDim.x + threadIdx.x) + start_at;
    if (idx<(partitions*partitions*partitions)){
        //Get the node positon of this thread
        int mom = (int) (idx/(partitions*partitions));
        int col = (int) ((idx%(partitions*partitions))/partitions);
        int row = idx%partitions;
        
        if (nodeD[row][col][mom].len > 0){

            float ds = ((float)(bn))/dmax, dd_max=dmax*dmax;
            float nx1=nodeD[row][col][mom].nodepos.x, ny1=nodeD[row][col][mom].nodepos.y, nz1=nodeD[row][col][mom].nodepos.z;
            float d_max_node = dmax + size_node*sqrt(3.0);
            d_max_node*=d_max_node;
            
            // Counts distances within the same node
            count_distances11(XX, nodeD[row][col][mom].elements, nodeD[row][col][mom].len, ds, dd_max, 2);
            
            
            int u=row,v=col,w=mom; // Position index of the second node
            float dx_nod12, dy_nod12, dz_nod12, dd_nod12; //Internodal distance

            //Second node mobil in Z direction
            for(w = mom+1; w<partitions; w++){
                dz_nod12 = nodeD[u][v][w].nodepos.z - nz1;
                dd_nod12 = dz_nod12*dz_nod12;
                if (dd_nod12 <= d_max_node){
                    count_distances12(XX, nodeD[row][col][mom].elements, nodeD[row][col][mom].len, nodeD[row][col][w].elements, nodeD[row][col][w].len, ds, dd_max, 2);
                }
            }

            //Second node mobil in YZ
            for(v=col+1; v<partitions; v++){
                dy_nod12 = nodeD[u][v][0].nodepos.y - ny1;
                for(w=0; w<partitions; w++){
                    dz_nod12 = nodeD[u][v][w].nodepos.z - nz1;
                    dd_nod12 = dz_nod12*dz_nod12 + dy_nod12*dy_nod12;
                    if (dd_nod12<=d_max_node){
                        count_distances12(XX, nodeD[row][col][mom].elements, nodeD[row][col][mom].len, nodeD[row][v][w].elements, nodeD[row][v][w].len, ds, dd_max, 2);
                    }
                }
            }

            //Second node mobil in XYZ
            for(u = row+1; u < partitions; u++){
                dx_nod12 = nodeD[u][0][0].nodepos.x - nx1;
                for(v = 0; v < partitions; v++){
                    dy_nod12 = nodeD[u][v][0].nodepos.y - ny1;
                    for(w = 0; w < partitions; w++){
                        dz_nod12 = nodeD[u][v][w].nodepos.z - nz1;
                        dd_nod12 = dz_nod12*dz_nod12 + dy_nod12*dy_nod12 + dx_nod12*dx_nod12;
                        if (dd_nod12<=d_max_node){
                            count_distances12(XX, nodeD[row][col][mom].elements, nodeD[row][col][mom].len, nodeD[u][v][w].elements, nodeD[u][v][w].len, ds, dd_max, 2);
                        }
                    }
                }
            }
            
        }
    }
}
__global__ void make_histoXY(float *XY, Node ***nodeD, Node ***nodeR, int partitions, int bn, float dmax, float size_node, int start_at){
    int idx = 2*(blockIdx.x * blockDim.x + threadIdx.x) + start_at;
    if (idx<(partitions*partitions*partitions)){
        //Get the node positon in this thread
        int mom = (int) (idx/(partitions*partitions));
        int col = (int) ((idx%(partitions*partitions))/partitions);
        int row = idx%partitions;
        
        if (nodeD[row][col][mom].len > 0){

            float ds = ((float)(bn))/dmax, dd_max=dmax*dmax;
            float nx1=nodeD[row][col][mom].nodepos.x, ny1=nodeD[row][col][mom].nodepos.y, nz1=nodeD[row][col][mom].nodepos.z;
            float d_max_node = dmax + size_node*sqrt(3.0);
            d_max_node*=d_max_node;
            
            int u,v,w; //Position of the second node
            unsigned int dx_nod12, dy_nod12, dz_nod12, dd_nod12;


            //Second node mobil in XYZ
            for(u = 0; u < partitions; u++){
                dx_nod12 = nodeD[u][0][0].nodepos.x - nx1;
                for(v = 0; v < partitions; v++){
                    dy_nod12 = nodeD[u][v][0].nodepos.y - ny1;
                    for(w = 0; w < partitions; w++){
                        dz_nod12 = nodeD[u][v][w].nodepos.z - nz1;
                        dd_nod12 = dz_nod12*dz_nod12 + dy_nod12*dy_nod12 + dx_nod12*dx_nod12;
                        if (dd_nod12<=d_max_node){
                            count_distances12(XY, nodeD[row][col][mom].elements, nodeD[row][col][mom].len, nodeR[u][v][w].elements, nodeR[u][v][w].len, ds, dd_max, 1);
                        }
                    }
                }
            }
            
        }
    }
}

int main(int argc, char **argv){
	
    int np = stoi(argv[3]), bn = stoi(argv[4]);
    float dmax; // = stof(argv[5]);

    float *DD_A, *RR_A, *DR_A, *DD_B, *RR_B, *DR_B;
    double *DD, *RR, *DR;
    PointW3D *dataD;
    PointW3D *dataR;
    hipMallocManaged(&dataD, np*sizeof(PointW3D));
    hipMallocManaged(&dataR, np*sizeof(PointW3D));

    // Name of the files where the results are saved
    string nameDD = "DDiso.dat", nameRR = "RRiso.dat", nameDR = "DRiso.dat";

    // Allocate memory for the histogram as double
    // And the subhistograms as simple presision floats
    DD = new double[bn];
    RR = new double[bn];
    DR = new double[bn];
    hipMallocManaged(&DD_A, bn*sizeof(float));
    hipMallocManaged(&RR_A, bn*sizeof(float));
    hipMallocManaged(&DR_A, bn*sizeof(float));
    hipMallocManaged(&DD_B, bn*sizeof(float));
    hipMallocManaged(&RR_B, bn*sizeof(float));
    hipMallocManaged(&DR_B, bn*sizeof(float));
	
    // Open and read the files to store the data in the arrays
    float size_box = 0; //, dummy_size_box=0; //Will be obtained from the data
    open_files(argv[1], np, dataD, size_box);
    cout << "Successfully readed the file "<< argv[1] << endl;
    cout << "Size of the box of " << size_box << endl; 
    //open_files(argv[2], np, dataR, dummy_size_box);
    
    float size_node;//, prev_time = numeric_limits<double>::infinity();// = 2.176;
    //float size_node = alpha*(size_box/pow((float)(np),1/3.));
    int partitions;// = (int)(ceil(size_box/size_node));
    ofstream outfile; //To write optimization results

    for (dmax=20; dmax<140; dmax=dmax+10){
        cout << "Computing for a dmax of: "<< dmax <<endl;

        for (partitions=10; partitions<100; partitions+=5){
            size_node = size_box/((float)(partitions));
            cout << "Trying with " << partitions << " partitions and a size node of " << size_node << endl;
            //Initialize the histograms in 0
            for (int i = 0; i < bn; i++){
                *(DD+i) = 0;
                *(RR+i) = 0;
                *(DR+i) = 0;
                *(DD_A+i) = 0;
                *(RR_A+i) = 0;
                *(DR_A+i) = 0;
                *(DD_B+i) = 0;
                *(RR_B+i) = 0;
                *(DR_B+i) = 0;
            }

            //Init the nodes arrays
            Node ***nodeD;
            //Node ***nodeR;
            //cudaMallocManaged(&nodeR, partitions*sizeof(Node**));
            hipMallocManaged(&nodeD, partitions*sizeof(Node**));
            for (int i=0; i<partitions; i++){
                //cudaMallocManaged(&*(nodeR+i), partitions*sizeof(Node*));
                hipMallocManaged(&*(nodeD+i), partitions*sizeof(Node*));
                for (int j=0; j<partitions; j++){
                    //cudaMallocManaged(&*(*(nodeR+i)+j), partitions*sizeof(Node));
                    hipMallocManaged(&*(*(nodeD+i)+j), partitions*sizeof(Node));
                }
            }

            
            //Classificate the data into the nodes
            make_nodos(nodeD, dataD, partitions, size_node, np);
            //make_nodos(nodeR, dataR, partitions, size_node, np);

            //Get the dimensions of the GPU grid
            int threads = 512;
            int blocks = (int)(ceil((float)((partitions*partitions*partitions)/(float)(2*threads))));
            dim3 grid(blocks,1,1);
            dim3 block(threads,1,1);
            //One thread for each node

            clock_t begin = clock();
            //Launch the kernels
            make_histoXX<<<grid,block>>>(DD_A, nodeD, partitions, bn, dmax, size_node, 0);
            make_histoXX<<<grid,block>>>(DD_B, nodeD, partitions, bn, dmax, size_node, 1);
            //make_histoXX<<<grid,block>>>(RR_A, nodeR, partitions, bn, dmax, size_node, 0);
            //make_histoXX<<<grid,block>>>(RR_B, nodeR, partitions, bn, dmax, size_node, 1);
            //make_histoXY<<<grid,block>>>(DR_A, nodeD, nodeR, partitions, bn, dmax, size_node, 0);
            //make_histoXY<<<grid,block>>>(DR_B, nodeD, nodeR, partitions, bn, dmax, size_node, 1);

            //Waits for the GPU to finish
            hipDeviceSynchronize();  

            //Check here for errors
            hipError_t error = hipGetLastError(); 
            cout << "The error code is " << error << endl;
            if(error != 0)
            {
            // print the CUDA error message and exit
            printf("CUDA error: %s\n", hipGetErrorString(error));
            exit(-1);
            }

            clock_t end = clock();
            double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
            printf("\nSpent time = %.4f seg.\n", time_spent );

            //"time_results.dat"
            //n_points, size_box, dmax, partitions, node_size, time [s]
            outfile.open("time_results.dat", ios_base::app); // append instead of overwrite
            outfile << np << ", " << size_box << ", " << dmax << ", " << partitions << ", " << size_node << ", " <<  time_spent << endl;
            outfile.close();

            //Collect the subhistograms data into the double precision main histograms
            //THis has to be done in CPU since GPU only allows single precision
            for (int i = 0; i < bn; i++){
                DD[i] = (double)(DD_A[i]+ DD_B[i]);
                RR[i] = (double)(RR_A[i]+ RR_B[i]);
                DR[i] = (double)(DR_A[i]+ DR_B[i]);
            }


            for (int i=0; i<partitions; i++){
                for (int j=0; j<partitions; j++){
                    //cudaFree(&*(*(nodeR+i)+j));
                    hipFree(*(*(nodeD+i)+j));
                }
                //cudaFree(&*(nodeR+i));
                hipFree(*(nodeD+i));
            }
            //cudaFree(&nodeR);
            hipFree(nodeD);

            //Check here for errors
            error = hipGetLastError(); 
            cout << "The error code is " << error << endl;
        }

    }

    /*
    cout << "Termine de hacer todos los histogramas" << endl;
	
	// Guardamos los histogramas
	save_histogram(nameDD, bn, DD);
	cout << "Guarde histograma DD..." << endl;
	save_histogram(nameRR, bn, RR);
	cout << "Guarde histograma RR..." << endl;
	save_histogram(nameDR, bn, DR);
    cout << "Guarde histograma DR..." << endl;
    */

    //Free the memory
    hipFree(dataD);
    hipFree(dataR);

    delete[] DD;
    delete[] DR;
    delete[] RR;
    hipFree(DD_A);
    hipFree(RR_A);
    hipFree(DR_A);
    hipFree(DD_B);
    hipFree(RR_B);
    hipFree(DR_B);

    cout << "Programa Terminado..." << endl;
    return 0;
}

